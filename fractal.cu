#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include "opencv2/opencv.hpp"

struct Vector2f
{
	float x = 0;
	float y = 0;

	__host__ __device__
		float norm()
	{
		return std::sqrt(x * x + y * y);
	}
};

__host__ __device__
Vector2f operator+(Vector2f a, Vector2f b)
{
	return Vector2f{ a.x + b.x ,a.y + b.y };
}

__host__ __device__
Vector2f complex_sqr(Vector2f z)
{
	return Vector2f{ z.x * z.x - z.y * z.y, z.x * z.y * 2 };
}

__global__
void Fractal(int n, float t, float* pixels)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	Vector2f c = Vector2f{ -0.8,cos(t) * 0.2 };
	Vector2f z = Vector2f{ (double)i / n - 1,(double)j / n - 0.5 };
	z.x = z.x * 2;
	z.y = z.y * 2;

	int iterator = 0;
	while ((z.norm() < 20) && (iterator < 50))
	{
		z = complex_sqr(z) + c;
		iterator++;
	}
	//printf("%d\n", iterator);
	pixels[j * 2 * n + i] = 1 - iterator * 0.02;
}



int main(int argc, char* argv[])
{

	int n = 320;
	float t = 10000;

	float* pixels;
	hipMallocManaged((void**)&pixels, 2 * n * n * sizeof(float));

	dim3 gridSize(20, 10);
	dim3 blockSize(32, 32);

	cv::Mat img(n, 2 * n, CV_32FC1);

	// ��ʱ��ʼ
	hipEvent_t start_gpu = 0, stop_gpu = 0;
	hipEventCreate(&start_gpu);
	hipEventCreate(&stop_gpu);

	for (int i = 0; i < 1000000; i++)
	{

		hipEventRecord(start_gpu);
		Fractal << <gridSize, blockSize >> > (n, i * 0.03, pixels);
		//hipDeviceSynchronize();
		hipEventRecord(stop_gpu);

		hipEventSynchronize(start_gpu);
		hipEventSynchronize(stop_gpu);
		float gpu_time = 0;
		hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);
		std::cout << 1000.0 / gpu_time << " fps" << std::endl;

		memcpy(img.data, pixels, 2 * n * n * sizeof(float));

		cv::imshow("res", img);
		cv::waitKey(1);


	}

	hipEventDestroy(start_gpu);
	hipEventDestroy(stop_gpu);

	return 0;
}